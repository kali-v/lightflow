
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <iostream>

#ifdef LF_CUDA_AVAIL
#include "cuda_runtime.h"
#endif

void move_data_to_cuda(const float* host_ptr, const int size, float** dev_ptr) {
    hipMalloc(dev_ptr, size * sizeof(float));
    hipMemcpy(*dev_ptr, host_ptr, size * sizeof(float), hipMemcpyHostToDevice);
}

void move_data_to_host(float* host_ptr, const float* dev_ptr, const int size) {
    hipMemcpy(host_ptr, dev_ptr, size * sizeof(float), hipMemcpyDeviceToHost);
}

__global__ void add_kernel(const float* a, const float* b, float* c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

__global__ void sub_kernel(const float* a, const float* b, float* c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = a[i] - b[i];
    }
}

__global__ void mul_kernel(const float* a, const float* b, float* c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = a[i] * b[i];
    }
}

__global__ void div_kernel(const float* a, const float* b, float* c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = a[i] / b[i];
    }
}

__global__ void pow_kernel(const float* a, const float* exp, float* c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = powf(a[i], exp[0]);
    }
}

__global__ void pow_const_kernel(const float* a, const float exp, float* c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = powf(a[i], exp);
    }
}

__global__ void compare_arrays_kernel(const float* a, const float* b, float* res, const float threshold,
                                      const int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        if (fabs(a[i] - b[i]) > threshold) atomicExch(res, 1.0f);
    }
}

__global__ void matmul_cuda_kernel(const float* a, const float* b, float* c, const int ah, const int aw, const int bw) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (j < bw && i < ah) {
        float tmp = 0;
        for (int k = 0; k < aw; k++) {
            tmp += a[i * aw + k] * b[k * bw + j];
        }
        c[i * bw + j] = tmp;
    }
}

__global__ void matmul_deep_cuda_kernel(const float* a, const float* b, float* c, const int ah, const int aw,
                                        const int bw, const int ch, const int bs, const int a_size, const int b_size,
                                        const int c_size) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    for (int b0 = 0; b0 < bs; b0++) {
        for (int b1 = 0; b1 < ch; b1++) {
            int tof = b0 * ch * a_size + b1 * a_size;
            int oof = b0 * ch * b_size + b1 * b_size;
            int rof = b0 * ch * c_size + b1 * c_size;

            if (j < bw && i < ah) {
                float tmp = 0;
                for (int k = 0; k < aw; k++) {
                    tmp += a[tof + i * aw + k] * b[oof + k * bw + j];
                }
                c[rof + i * bw + j] = tmp;
            }
        }
    }
}

void add_cuda(const float* a, const float* b, float* c, int size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    add_kernel<<<num_blocks, block_size>>>(a, b, c, size);
}

void sub_cuda(const float* a, const float* b, float* c, int size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    sub_kernel<<<num_blocks, block_size>>>(a, b, c, size);
}

void mul_cuda(const float* a, const float* b, float* c, int size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    mul_kernel<<<num_blocks, block_size>>>(a, b, c, size);
}

void div_cuda(const float* a, const float* b, float* c, int size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    div_kernel<<<num_blocks, block_size>>>(a, b, c, size);
}

void pow_const_cuda(const float* a, const float exp, float* c, int size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    pow_const_kernel<<<num_blocks, block_size>>>(a, exp, c, size);
}

void pow_cuda(const float* a, const float* exp, float* c, int size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    pow_kernel<<<num_blocks, block_size>>>(a, exp, c, size);
}

bool compare_arrays_cuda(const float* a, const float* b, const float threshold, const int size) {
    float* res;
    hipMallocManaged(&res, sizeof(float));
    res[0] = 0.0;

    const int blocks = 256;
    const int grids = (size + blocks - 1) / blocks;
    compare_arrays_kernel<<<grids, blocks>>>(a, b, res, threshold, size);
    hipDeviceSynchronize();

    bool result = (*res == 0.0);
    hipFree(res);

    return result;
}

void matmul_cuda(const float* a, const float* b, float* res, const int ah, const int aw, const int bw) {
    int bs = 32;
    dim3 grids(std::ceil(bw / (float)bs), std::ceil(ah / (float)bs));
    dim3 blocks(bs, bs);

    matmul_cuda_kernel<<<grids, blocks>>>(a, b, res, ah, aw, bw);
}

void matmul_deep_cuda(const float* a, const float* b, float* res, const int ah, const int aw, const int bw,
                      const int ch, const int bs) {
    int block_size = 16;
    dim3 grids(std::ceil(bw / (float)block_size), std::ceil(ah / (float)block_size));
    dim3 blocks(block_size, block_size);

    int a_size = ah * aw;
    int b_size = aw * bw;
    int c_size = ah * bw;

    matmul_deep_cuda_kernel<<<grids, blocks>>>(a, b, res, ah, aw, bw, ch, bs, a_size, b_size, c_size);
}
