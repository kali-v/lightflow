#include "stdio.h"
#include <iostream>

#include "hip/hip_runtime.h"

__global__ void compare_arrays_kernel(float* a, float* b, float* res, float threshold, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        if (fabs(a[i] - b[i]) > threshold)
            atomicExch(res, 1.0f);
    }
}

__global__ void matmul_cuda_kernel(float* a, float* b, float* c, int ah, int aw, int bw) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (j < bw && i < ah) {
        float tmp = 0;
        for (int k = 0; k < aw; k++) {
            tmp += a[i * aw + k] * b[k * bw + j];
        }
        c[i * bw + j] = tmp;
    }
}

__global__ void matmul_deep_cuda_kernel(float* a, float* b, float* c, int ah, int aw, int bw, int ch, int bs,
                                        int a_size, int b_size, int c_size) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    for (int b0 = 0; b0 < bs; b0++) {
        for (int b1 = 0; b1 < ch; b1++) {
            int tof = b0 * ch * a_size + b1 * a_size;
            int oof = b0 * ch * b_size + b1 * b_size;
            int rof = b0 * ch * c_size + b1 * c_size;

            if (j < bw && i < ah) {
                float tmp = 0;
                for (int k = 0; k < aw; k++) {
                    tmp += a[tof + i * aw + k] * b[oof + k * bw + j];
                }
                c[rof + i * bw + j] = tmp;
            }
        }
    }
}

bool compare_arrays_cuda(float* a, float* b, float threshold, int size) {
    float* res;
    hipMallocManaged(&res, sizeof(float));
    res[0] = 0.0;

    const int blocks = 256;
    const int grids = (size + blocks - 1) / blocks;
    compare_arrays_kernel<<<grids, blocks>>>(a, b, res, threshold, size);
    hipDeviceSynchronize();

    bool result = (*res == 0.0);
    hipFree(res);

    return result;
}

void matmul_cuda(float* a, float* b, float* res, int ah, int aw, int bw) {
    int bs = 32;
    dim3 grids(std::ceil(bw / (float)bs), std::ceil(ah / (float)bs));
    dim3 blocks(bs, bs);

    matmul_cuda_kernel<<<grids, blocks>>>(a, b, res, ah, aw, bw);
}

void matmul_deep_cuda(float* a, float* b, float* res, int ah, int aw, int bw, int ch, int bs) {
    int block_size = 16;
    dim3 grids(std::ceil(bw / (float)block_size), std::ceil(ah / (float)block_size));
    dim3 blocks(block_size, block_size);

    int a_size = ah * aw;
    int b_size = aw * bw;
    int c_size = ah * bw;

    matmul_deep_cuda_kernel<<<grids, blocks>>>(a, b, res, ah, aw, bw, ch, bs, a_size, b_size, c_size);
}
