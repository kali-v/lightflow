
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <iostream>

#ifdef LF_CUDA_AVAIL
#include "cuda_runtime.h"
#endif

void move_data_to_cuda(const float* host_ptr, const int size, float** dev_ptr) {
    hipMalloc(dev_ptr, size * sizeof(float));
    hipMemcpy(*dev_ptr, host_ptr, size * sizeof(float), hipMemcpyHostToDevice);
}

void move_data_to_host(float* host_ptr, const float* dev_ptr, const int size) {
    hipMemcpy(host_ptr, dev_ptr, size * sizeof(float), hipMemcpyDeviceToHost);
}

__global__ void add_kernel(const float* a, const float* b, float* c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

__global__ void add_const_kernel(const float* a, const float* b, float* c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = a[i] + b[0];
    }
}

__global__ void sub_kernel(const float* a, const float* b, float* c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = a[i] - b[i];
    }
}
__global__ void sub_const_kernel(const float* a, const float* b, float* c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = a[i] - b[0];
    }
}

__global__ void mul_kernel(const float* a, const float* b, float* c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = a[i] * b[i];
    }
}
__global__ void mul_const_kernel(const float* a, const float* b, float* c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = a[i] * b[0];
    }
}

__global__ void div_kernel(const float* a, const float* b, float* c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = a[i] / b[i];
    }
}
__global__ void div_const_kernel(const float* a, const float* b, float* c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = a[i] / b[0];
    }
}

__global__ void sqrt_kernel(const float* a, float* b, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        b[i] = sqrtf(a[i]);
    }
}

__global__ void exp_kernel(const float* a, float* b, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        b[i] = __expf(a[i]);
    }
}
__global__ void relu_kernel(const float* a, float* b, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        b[i] = a[i] > 0 ? a[i] : 0;
    }
}
__global__ void log_kernel(const float* a, float* b, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        b[i] = logf(a[i]);
    }
}

__global__ void pow_kernel(const float* a, const float* exp, float* c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = powf(a[i], exp[0]);
    }
}

__global__ void pow_const_kernel(const float* a, const float exp, float* c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = powf(a[i], exp);
    }
}

__global__ void compare_arrays_kernel(const float* a, const float* b, float* res, const float threshold,
                                      const int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        if (fabs(a[i] - b[i]) > threshold) atomicExch(res, 1.0f);
    }
}

__global__ void matmul_cuda_kernel(const float* a, const float* b, float* c, const int ah, const int aw, const int bw) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (j < bw && i < ah) {
        float tmp = 0;
        for (int k = 0; k < aw; k++) {
            tmp += a[i * aw + k] * b[k * bw + j];
        }
        c[i * bw + j] = tmp;
    }
}

__global__ void matmul_deep_cuda_kernel(const float* a, const float* b, float* c, const int ah, const int aw,
                                        const int bw, const int ch, const int bs, const int a_size, const int b_size,
                                        const int c_size) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    for (int b0 = 0; b0 < bs; b0++) {
        for (int b1 = 0; b1 < ch; b1++) {
            int tof = b0 * ch * a_size + b1 * a_size;
            int oof = b0 * ch * b_size + b1 * b_size;
            int rof = b0 * ch * c_size + b1 * c_size;

            if (j < bw && i < ah) {
                float tmp = 0;
                for (int k = 0; k < aw; k++) {
                    tmp += a[tof + i * aw + k] * b[oof + k * bw + j];
                }
                c[rof + i * bw + j] = tmp;
            }
        }
    }
}

__global__ void transpose_kernel(const float* a, float* res, int bs, int ch, int h, int w) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int n = tid; n < bs; n += stride) {
        for (int c = 0; c < ch; c++) {
            int tof = (n * ch + c) * h * w;
            for (int i = 0; i < h; i++) {
                for (int j = 0; j < w; j++) {
                    res[tof + j * h + i] = a[tof + i * w + j];
                }
            }
        }
    }
}

void add_cuda(const float* a, const float* b, float* c, int asize, int bsize) {
    int block_size = 256;
    int num_blocks = (asize + block_size - 1) / block_size;
    if (bsize == 1)
        add_const_kernel<<<num_blocks, block_size>>>(a, b, c, asize);
    else
        add_kernel<<<num_blocks, block_size>>>(a, b, c, asize);
}

void sub_cuda(const float* a, const float* b, float* c, int asize, int bsize) {
    int block_size = 256;
    int num_blocks = (asize + block_size - 1) / block_size;
    if (bsize == 1)
        sub_const_kernel<<<num_blocks, block_size>>>(a, b, c, asize);
    else
        sub_kernel<<<num_blocks, block_size>>>(a, b, c, asize);
}

void mul_cuda(const float* a, const float* b, float* c, int asize, int bsize) {
    int block_size = 256;
    int num_blocks = (asize + block_size - 1) / block_size;
    if (bsize == 1)
        mul_const_kernel<<<num_blocks, block_size>>>(a, b, c, asize);
    else
        mul_kernel<<<num_blocks, block_size>>>(a, b, c, asize);
}

void div_cuda(const float* a, const float* b, float* c, int asize, int bsize) {
    int block_size = 256;
    int num_blocks = (asize + block_size - 1) / block_size;
    if (bsize == 1)
        div_const_kernel<<<num_blocks, block_size>>>(a, b, c, asize);
    else
        div_kernel<<<num_blocks, block_size>>>(a, b, c, asize);
}

void pow_const_cuda(const float* a, const float exp, float* c, int size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    pow_const_kernel<<<num_blocks, block_size>>>(a, exp, c, size);
}

void pow_cuda(const float* a, const float* exp, float* c, int size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    pow_kernel<<<num_blocks, block_size>>>(a, exp, c, size);
}

void sqrt_cuda(const float* a, float* b, int size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    sqrt_kernel<<<num_blocks, block_size>>>(a, b, size);
}

void log_cuda(const float* a, float* b, int size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    log_kernel<<<num_blocks, block_size>>>(a, b, size);
}

void exp_cuda(const float* a, float* b, int size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    exp_kernel<<<num_blocks, block_size>>>(a, b, size);
}

void relu_cuda(const float* a, float* b, int size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    relu_kernel<<<num_blocks, block_size>>>(a, b, size);
}

bool compare_arrays_cuda(const float* a, const float* b, const float threshold, const int size) {
    float* res;
    hipMallocManaged(&res, sizeof(float));
    res[0] = 0.0;

    const int blocks = 256;
    const int grids = (size + blocks - 1) / blocks;
    compare_arrays_kernel<<<grids, blocks>>>(a, b, res, threshold, size);
    hipDeviceSynchronize();

    bool result = (*res == 0.0);
    hipFree(res);

    return result;
}

void matmul_cuda(const float* a, const float* b, float* res, const int ah, const int aw, const int bw) {
    int bs = 32;
    dim3 grids(std::ceil(bw / (float)bs), std::ceil(ah / (float)bs));
    dim3 blocks(bs, bs);

    matmul_cuda_kernel<<<grids, blocks>>>(a, b, res, ah, aw, bw);
}

void matmul_deep_cuda(const float* a, const float* b, float* res, const int ah, const int aw, const int bw,
                      const int ch, const int bs) {
    int block_size = 16;
    dim3 grids(std::ceil(bw / (float)block_size), std::ceil(ah / (float)block_size));
    dim3 blocks(block_size, block_size);

    int a_size = ah * aw;
    int b_size = aw * bw;
    int c_size = ah * bw;

    matmul_deep_cuda_kernel<<<grids, blocks>>>(a, b, res, ah, aw, bw, ch, bs, a_size, b_size, c_size);
}

void transpose_cuda(const float* a, float* res, const int bs, const int ch, const int h, const int w) {
    int threads_per_block = 256;
    int num_blocks = (bs + threads_per_block - 1) / threads_per_block;

    transpose_kernel<<<num_blocks, threads_per_block>>>(a, res, bs, ch, h, w);
}
